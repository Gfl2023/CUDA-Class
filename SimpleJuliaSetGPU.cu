#include "hip/hip_runtime.h"
//nvcc SimpleJuliaSetGPU.cu -o SimpleJuliaSetGPU -lglut -lGL -lm
// This is a simple Julia set which is repeated iterations of 
// Znew = Zold + C whre Z and Care imaginary numbers.
// After so many tries if Zinitial escapes color it black if it stays around color it red.

#include <GL/glut.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "./CUDAERROR.h"


#define A  -0.8789  //real
#define B  -0.11234009   //imaginary
#define N 1024

unsigned int window_width = 1024;
unsigned int window_height = 1024;

float xMin = -2.0;
float xMax =  2.0;
float yMin = -2.0;
float yMax =  2.0;
dim3 BlockSize;
dim3 GridSize;
float stepSizeX = (xMax - xMin)/((float)window_width);
float stepSizeY = (yMax - yMin)/((float)window_height);
float *pixels;
float *pixels_GPU; 

void SetUpCudaDevices()
{
	BlockSize.x = 1024;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	GridSize.x = 1024;
	GridSize.y = 1;
	GridSize.z = 1;
}

void AllocateMemory()
{					
	//Allocate Device (GPU) Memory
	hipMalloc(&pixels_GPU,N*3*N*sizeof(float));
	
	//Allocate Host (CPU) Memory
	pixels = (float*)malloc(N*3*N*sizeof(float));

	
	
	
}

__device__ float color (float x, float y) 
{
	float mag,maxMag,temp;
	float maxCount = 200;
	float count = 0;
	maxMag = 10;
	mag = 0.0;
	
	while (mag < maxMag && count < maxCount) 
	{
		// Zn = Zo*Zo + C
		// or xn + yni = (xo + yoi)*(xo + yoi) + A + Bi
		// xn = xo*xo - yo*yo + A (real Part) and yn = 2*xo*yo + B (imagenary part)
		temp = x; // We will be changing the x but weneed its old value to hind y.	
		x = x*x - y*y + A;
		y = (2.0 * temp * y) + B;
		mag = sqrt(x*x + y*y);
		count++;
	}
	if(count < maxCount) 
	{
		return(0.0);
	}
	else
	{
		return(1.0);
	}
}

__global__ void initialization(float *g, float stepSizeX, float stepSizeY, float xmin, float ymin) 
{ 
	float x,y;

	int id = (blockDim.x*blockIdx.x + threadIdx.x)*3;
	y = ymin + blockIdx.x*stepSizeY;
	x = xmin + threadIdx.x*stepSizeX;

	g[id] = color(x,y); //red 1.0 on 0.0 off
	g[id=1] = 0.0; // blue off
	g[id=2] = 0.0; // green off

	
}

void display(void)
{
	 initialization<<<GridSize,BlockSize>>>(pixels_GPU,stepSizeX,stepSizeY,xMin,yMin);
	 hipMemcpyAsync(pixels,pixels_GPU,N*3*N*sizeof(float),hipMemcpyDeviceToHost);
	 errorCheck(__FILE__, __LINE__);
	 glDrawPixels(window_width,window_height, GL_RGB, GL_FLOAT, pixels);
	 glFlush();
}

int main(int argc, char** argv)
{ 
	
	SetUpCudaDevices();
	AllocateMemory();
	hipMemcpyAsync(pixels_GPU,pixels,N*3*N*sizeof(float),hipMemcpyHostToDevice);
	errorCheck(__FILE__, __LINE__);
   	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
   	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Fractals man, fractals.");
   	glutDisplayFunc(display);
   	glutMainLoop();
}
